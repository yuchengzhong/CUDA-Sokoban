#include "hip/hip_runtime.h"
#include "solver_gpu_global.cuh"

// Globals
__global__ void GenerateSolverStates(const ATOMIC_SolverState* d_SolverStates, int StatesSize, ATOMIC_SolverState* d_NewSolverStates)
{
    int t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t < StatesSize)
    {
        for (int i = 0; i < 4; i++)
        {
            ATOMIC_SolverState Candidate = d_SolverStates[t];
            int2 CurrentMoveStep = ATOMIC_Steps::GetStepByIndex(i);

            bool bMoveValid = Candidate.SceneState.MovePlayer(CurrentMoveStep);
            Candidate.SceneState.UpdatePhysics();
            Candidate.StepState.AddStep(CurrentMoveStep);
            Candidate.WinState = Candidate.SceneState.bIsWin();
            Candidate.ValidState = bMoveValid;
            d_NewSolverStates[i + 4 * t] = Candidate;
        }
    }
}
#define CHUNK_SIZE 32
__global__ void MarkInvalidDuplicatesFromGlobal(ATOMIC_SolverState* StatesToMark, int N_StatesToMark, ATOMIC_SolverState* StatesFind, int N_StatesFind)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ Actor SHARED_ActorsFind[CHUNK_SIZE * ATOMIC_MAX_ACTORS];
    __shared__ int SHARED_ActorCount;


    if (threadIdx.x == 0)
    {
        SHARED_ActorCount = StatesToMark[0].SceneState.ActorCount;
    }
    __syncthreads();

    if (idx >= N_StatesToMark)
    {
        return;
    }

    int N_Chunks = (N_StatesFind + CHUNK_SIZE - 1) / CHUNK_SIZE;

    for (int Chunk = 0; Chunk < N_Chunks; Chunk++)
    {
        int ChunkIdx = Chunk * CHUNK_SIZE + threadIdx.x;
        if (ChunkIdx < N_StatesFind)
        {
            for (int j = 0; j < SHARED_ActorCount; ++j)
            {
                SHARED_ActorsFind[threadIdx.x * SHARED_ActorCount + j] = StatesFind[ChunkIdx].SceneState.Actors[j];
            }
        }
        __syncthreads();
        for (int i = 0; i < CHUNK_SIZE && (Chunk * CHUNK_SIZE + i) < N_StatesFind; i++)
        {
            bool bCanSkip = false;
            for (int j = 0; j < SHARED_ActorCount; ++j)
            {
                if (StatesToMark[idx].SceneState.Actors[j] != SHARED_ActorsFind[i * SHARED_ActorCount + j])
                {
                    bCanSkip = true;
                    break;
                }
            }
            if (bCanSkip)
            {
                break;
            }
            StatesToMark[idx].ValidState = false;
        }
        __syncthreads();
    }
}
#undef CHUNK_SIZE
